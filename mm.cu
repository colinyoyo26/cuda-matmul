#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <time.h>

#define MAXN 8192
#define MAXMAT MAXN *MAXN

#ifdef TILING
#define BLKW 16
#define BLKH BLKW
#define BLKDIM dim3(BLKW, BLKH)
#define GRIDDIM dim3(N / BLKW, N / BLKH)
#define TILEN BLKW
__global__ void matmul(uint32_t A[], uint32_t B[], uint32_t C[], uint32_t N)
{
    int blkc = blockIdx.x, blkr = blockIdx.y;
    int tidc = threadIdx.x, tidr = threadIdx.y; 
    int row = blkr * TILEN + tidr, col = blkc * TILEN + tidc;
    __shared__ uint32_t packA[TILEN][TILEN], packB[TILEN][TILEN];
    uint32_t sum = 0; 
    for (int i = 0; i < N / TILEN; i++) { 
        packA[tidr][tidc] = A[row * N + i * TILEN + tidc];
        packB[tidr][tidc] = B[(i * TILEN + tidr) * N + col];
        __syncthreads();
#pragma unroll
        for (int j = 0; j < TILEN; j++)
            sum += packA[tidr][j] * packB[j][tidc];
        __syncthreads();
    }
    C[row * N + col] = sum;
}

#elif ELEPACK
#define BLKSZ 64
#define BLKDIM dim3(BLKSZ)
#define GRIDDIM dim3(N *N / BLKSZ)
__global__ void matmul(uint32_t A[], uint32_t B[], uint32_t C[], uint32_t N)
{
    int tid = threadIdx.x, blksz = blockDim.x, idx = blockIdx.x * blksz + tid;
    int row = idx / N, col = idx % N;
    __shared__ uint32_t packA[MAXN];
    for (int i = tid; i < N; i += blksz)
        packA[i] = A[row * N + i];
    __syncthreads();
    uint32_t sum = 0; 
    for (int i = 0; i < N; i++) 
        sum += packA[i] * B[i * N + col];
    C[idx] = sum;
}

#elif ELE
#define BLKSZ 256
#define BLKDIM dim3(BLKSZ)
#define GRIDDIM dim3(N *N / BLKSZ)
__global__ void matmul(uint32_t A[], uint32_t B[], uint32_t C[], uint32_t N)
{
    int tid = threadIdx.x, blksz = blockDim.x, idx = blockIdx.x * blksz + tid;
    int row = idx / N, col = idx % N;
    uint32_t sum = 0; 
    for (int i = 0; i < N; i++) 
        sum += A[row * N + i] * B[i * N + col];
    C[idx] = sum;
}

#elif ROWPACK
#define BLKSZ 64
#define BLKDIM dim3(BLKSZ)
#define GRIDDIM dim3(N / BLKSZ)
__global__ void matmul(uint32_t A[], uint32_t B[], uint32_t C[], uint32_t N)
{
    int tid = threadIdx.x, blksz = blockDim.x, row = blockIdx.x * blksz + tid;
    __shared__ uint32_t packB[MAXN];
    for (int i = 0; i < N; i++) {
        for (int j = tid; j < N; j += blksz)
            packB[j] = B[j * N + i];
        __syncthreads();
        uint32_t sum = 0; 
        for (int j = 0; j < N; j++) 
            sum += A[row * N + j] * packB[j];
        C[row * N + i] = sum;
    }
}

#else
#define BLKSZ 64
#define BLKDIM dim3(BLKSZ)
#define GRIDDIM dim3(N / BLKSZ)
__global__ void matmul(uint32_t A[], uint32_t B[], uint32_t C[], uint32_t N)
{
    int tid = threadIdx.x, blksz = blockDim.x, row = blockIdx.x * blksz + tid;
    for (int i = 0; i < N; i++) {
        uint32_t sum = 0; 
        for (int j = 0; j < N; j++) 
            sum += A[row * N + j] * B[j * N + i];
        C[row * N + i] = sum;
    }
}
#endif

__global__ void print_sig(uint32_t A[], uint32_t N)
{
    uint32_t h = 0; 
    for (int i = 0; i < N * N; i++)
        h = (h + A[i]) * 2654435761LU;
    printf("Result: \n%u\n", h);
}

static inline void rand_gen(uint32_t c, int N, uint32_t A[])
{
    uint32_t x = 2, mod = N * N;
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++) { 
            x = (x * x + c + i + j) & (mod - 1);
            A[i * N + j] = x;
        }
}


static inline void init_all(uint32_t &N,
                            uint32_t &S1,
                            uint32_t &S2,
                            uint32_t A[],
                            uint32_t B[],
                            uint32_t **cuA,
                            uint32_t **cuB,
                            uint32_t **cuC)
{
#define SIZE (size_t)(N * N * sizeof(uint32_t))
    if (scanf("%u %u %u", &N, &S1, &S2) != 3)
        exit(1);
    rand_gen(S1, N, A);
    rand_gen(S2, N, B);
    hipMalloc(cuA, SIZE);
    hipMalloc(cuB, SIZE);
    hipMalloc(cuC, SIZE);
    hipMemcpy(*cuA, A, SIZE, hipMemcpyHostToDevice);
    hipMemcpy(*cuB, B, SIZE, hipMemcpyHostToDevice);
    hipMemset(*cuC, 0, SIZE);
#undef SIZE
}

static inline long get_time()
{
    struct timespec t;
    clock_gettime(CLOCK_REALTIME, &t);
    return (long) (t.tv_sec * 1e9 + t.tv_nsec);
}

static uint32_t A[MAXMAT], B[MAXMAT];
int main()
{
    uint32_t N, S1, S2, *cuA, *cuB, *cuC;
    init_all(N, S1, S2, A, B, &cuA, &cuB, &cuC);

    long t_start = get_time();
    matmul<<<GRIDDIM, BLKDIM>>>(cuA, cuB, cuC, N);
    long t_total = get_time() - t_start;
    print_sig<<<dim3(1), dim3(1)>>>(cuC, N);
    fprintf(stderr, "matmul time: \n%ld ns\n", t_total);
    hipFree(cuA);
    hipFree(cuB);
    hipFree(cuC);
    return 0;
}
